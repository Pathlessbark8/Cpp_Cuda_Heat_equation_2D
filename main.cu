
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<math.h>
#include<vector>
#include <chrono> 
#include <iomanip>
using namespace std::chrono; 
using namespace std;


// __global__ void Calculate(double *T,double *T_old,int r,int n)
// {
//     int j = blockDim.x*(blockIdx.x) + threadIdx.x;
//     int k = blockDim.y*(blockIdx.y) + threadIdx.y;
    
//     // printf("%d %d Hi\n",j,k);
//     if(j*n+k>n && j*n<=n*n)
//     {
        
//         *(T+j*n+k)=*(T_old+j*n+k)+r*( *(T_old+(j+1)*n+k)+*(T_old+j*n+k+1)+*(T_old+(j-1)*n+k)+*(T_old+j*n+k-1)- 4* *(T_old+j*n+k));
//         // T[j][k]= T_old[j][k]+r*(T_old[j+1][k]+ T_old[j][k+1]+ T_old[j-1][k]+ T_old[j][k-1]-4*T_old[j][k]);
//     }

// }


int main(){
    int n,ntime;
    double delta, sigma,nu,dom_len,dt,r;



    fstream fin;

    fin.open("input.dat",ios::in);

    fin>>n>>sigma>>nu>>dom_len>>ntime;

    fin.close();

    // cout<<n<<" "<<sigma<<" "<<nu<<" "<<dom_len<<" "<<ntime<<endl;;

    delta=dom_len/(n-1);

    dt=(sigma*pow(delta,2))/nu;



    double *T = (double *)malloc((n+1) * (n+1 )* sizeof(double));
    double *x = (double *)malloc((n+1) *( n+1) * sizeof(double));
    double *y = (double *)malloc((n+1) * (n+1) * sizeof(double));
    double *T_old = (double *)malloc((n+1) * (n+1) * sizeof(double));


auto start = high_resolution_clock::now(); 

// cout<<delta<<endl;
    for(int i=1;i<n+1;++i)
    {
        *(x+n+i)=0.0;
       
        *(x+n*n+i)=2.0;
      
        *(y+n*i+1)=0.0;
      
        *(y+n+n*i)=2.0;
    }
   

    for(int i=2;i<n;++i)
    {
        for(int j=1;j<n+1;++j)
        {    
            *(x+i*n+j)=*(x+(i-1)*n+j)+delta;
            *(y+j*n+i)=*(y+(j)*n+i-1)+delta;
            // cout<<i<<" "<<j<<" "<<x[i][j]<<endl;
        }
    }

   
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            if(*(x+i*n+j)<=1.5 && *(x+i*n+j)>=0.5 && *(y+i*n+j)<=1.5 && *(y+i*n+j) >=0.5)
            {
                *(T+i*n+j)=2.0;
            }           
            else{
                *(T+i*n+j)=1.0;
            }
        }
    }

    // for(int i=1;i<2;++i)
    // {
    //     for(int j=1;j<n+1;++j)
    //     {    
    //         cout<<*(y+i*n+j+1)<<" ";
    //     }
    //     cout<<endl;
    // }
    
    fstream foutw;
    foutw.open("int.dat",ios::out);
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            foutw<<*(x+i*n+j)<<" "<<*(y+i*n+j)<<" "<<*(T+i*n+j)<<"\n";
        }
    }
    foutw.close();
  
    r=(nu*dt)/pow(delta,2);

    

    // double*devPtr_T;
    // double*devPtr_T_old;
    // size_t pitch_T;
    // size_t pitch_T_old;
    // size_t host_pitch_T = n * sizeof(double);
    // size_t host_pitch_T_old = n * sizeof(double);


    


    // // cout << "sizeof(float): " << sizeof(double)<< endl;
    // // cout << "width: " << n << endl;
    // // cout << "height: " << n << endl;
    // // cout << "pitch:  " << pitch_T << endl;
    // // cout << "pitch:  " << pitch_T_old << endl;

  

    // cudaMallocPitch(&devPtr_T,&pitch_T,n * sizeof(double),n);
    // cudaMallocPitch(&devPtr_T_old,&pitch_T_old,n * sizeof(double),n);

    double *dev_T;
    double *dev_t_old;
  

    for(int i=1;i<=ntime;++i)
    {
        // cout<<"time_it:"<<i<<endl;
        for(int j=1;j<n+1;++j)
        {
            for(int k=1;k<n+1;++k)
            {
                *(T_old+j*n+k)=*(T+j*n+k);
            }
        }

        // cudaMemcpy2D(devPtr_T, pitch_T, &T, host_pitch_T, n*sizeof(double) , n, cudaMemcpyHostToDevice);
        // cudaMemcpy2D(devPtr_T_old, pitch_T_old,&T_old, host_pitch_T_old, n*sizeof(double) , n, cudaMemcpyHostToDevice);

        // dim3 a(32,32);
        // dim3 b(n/a.x,n/a.y);
        // Calculate<<<a,b>>>(devPtr_T,devPtr_T_old,r,n,pitch_T,pitch_T_old);
        // cudaDeviceSynchronize();

        // cudaMalloc(&dev_T,(n+1)*(n+1)*sizeof(double));
        // cudaMalloc(&dev_t_old,(n+1)*(n+1)*sizeof(double));
    
        // dim3 a(32,32);
        // dim3 b(n/a.x,n/a.y);
        // cudaMemcpy(dev_T, T, (n+1)*(n+1)*sizeof(double), cudaMemcpyHostToDevice);
        // cudaMemcpy(dev_t_old, T_old, (n+1)*(n+1)*sizeof(double), cudaMemcpyHostToDevice);

        // Calculate<<<a,b>>>(dev_T,dev_t_old,r,n);

        // cudaDeviceSynchronize();
        // cudaError_t error = cudaGetLastError();
        // if (error != cudaSuccess) {
        // fprintf(stderr, "ERROR: %s \n", cudaGetErrorString(error));
        // }


        // cudaMemcpy2D(T,host_pitch_T, devPtr_T, pitch_T, n*sizeof(double), n, cudaMemcpyDeviceToHost);
        // cudaMemcpy2D(T_old, host_pitch_T_old, devPtr_T_old, pitch_T_old, n*sizeof(double), n, cudaMemcpyDeviceToHost);


        for(int j=2;j<n;j++)
        {
            for(int k=2;k<n;k++)
            {
                *(T+j*n+k)=*(T_old+j*n+k)+r*( *(T_old+(j+1)*n+k)+*(T_old+j*n+k+1)+*(T_old+(j-1)*n+k)+*(T_old+j*n+k-1)- 4* *(T_old+j*n+k));
            }
        }
    }


    auto stop = high_resolution_clock::now(); 
    auto duration = duration_cast<milliseconds>(stop - start); 

    cout<<"simulation completed"<<endl;
    cout<<"Time_Taken : "<<duration.count()<<endl;

    fstream fout;
    fout.open("soln.dat",ios::out);
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            fout<<std::scientific<<*(x+i*n+j)<<" "<<*(y+i*n+j)<<" "<<*(T+i*n+j)<<"\n";
        }
    }
    fout.close();
    return 0;
}