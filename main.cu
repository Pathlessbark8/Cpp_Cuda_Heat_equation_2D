// #include<iostream>

#include <hip/hip_runtime.h>
#include<fstream>
#include<math.h>
#include<vector>
#include <chrono> 
#include <iomanip>
using namespace std::chrono; 
using namespace std;


__global__ void Calculate(double *T,double *T_old,int r,int n)
{
    int j = blockDim.x*(blockIdx.x) + threadIdx.x;
    int k = blockDim.y*(blockIdx.y) + threadIdx.y;

    if(j*n+k>n && j*n+k<=n*n)
    {
        
        *(T+j*n+k)=*(T_old+j*n+k)+r*( *(T_old+(j+1)*n+k)+*(T_old+j*n+k+1)+*(T_old+(j-1)*n+k)+*(T_old+j*n+k-1)- 4* *(T_old+j*n+k));
    }
    __syncthreads();
    
}


int main(){
    int n,ntime;
    double delta, sigma,nu,dom_len,dt,r;



    fstream fin;

    fin.open("input.dat",ios::in);

    fin>>n>>sigma>>nu>>dom_len>>ntime;

    fin.close();


    delta=dom_len/(n-1);

    dt=(sigma*pow(delta,2))/nu;



auto start = high_resolution_clock::now(); 

    double *T = (double *)malloc((n+1) * (n+1 )* sizeof(double));
    double *x = (double *)malloc((n+1) *( n+1) * sizeof(double));
    double *y = (double *)malloc((n+1) * (n+1) * sizeof(double));
    double *T_old = (double *)malloc((n+1) * (n+1) * sizeof(double));




    for(int i=1;i<n+1;++i)
    {
        *(x+n+i)=0.0;
       
        *(x+n*n+i)=2.0;
      
        *(y+n*i+1)=0.0;
      
        *(y+n+n*i)=2.0;
    }
   

    for(int i=2;i<n;++i)
    {
        for(int j=1;j<n+1;++j)
        {    
            *(x+i*n+j)=*(x+(i-1)*n+j)+delta;
            *(y+j*n+i)=*(y+(j)*n+i-1)+delta;
        }
    }

   
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            if(*(x+i*n+j)<=1.5 && *(x+i*n+j)>=0.5 && *(y+i*n+j)<=1.5 && *(y+i*n+j) >=0.5)
            {
                *(T+i*n+j)=2.0;
            }           
            else{
                *(T+i*n+j)=1.0;
            }
        }
    }


    
    fstream foutw;
    foutw.open("int.dat",ios::out);
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            foutw<<*(x+i*n+j)<<" "<<*(y+i*n+j)<<" "<<*(T+i*n+j)<<"\n";
        }
    }
    foutw.close();
  
    r=(nu*dt)/pow(delta,2);

    



    double *temp;
    double *dev_T;
    double *dev_t_old;


    
    hipMalloc(&dev_T,(n+1)*(n+1)*sizeof(double));
    hipMalloc(&dev_t_old,(n+1)*(n+1)*sizeof(double));
    hipMalloc(&temp,(n+1)*(n+1)*sizeof(double));

    // for(int j=1;j<n+1;++j)
    //     {
    //         for(int k=1;k<n+1;++k)
    //         {
    //             *(T_old+j*n+k)=*(T+j*n+k);
    //         }
    //     }
    
    hipMemcpy(dev_T, T, (n+1)*(n+1)*sizeof(double), hipMemcpyHostToDevice);

    for(int i=1;i<=ntime;++i)
    {
        //cout<<"time_it:"<<i<<endl;
        
       
        temp=dev_T;
        dev_T=dev_t_old;
        dev_t_old=temp;
        dim3 a(32,8);
        dim3 b(n/a.x,n/a.y);
        // cudaMemcpy(dev_t_old, T_old, (n+1)*(n+1)*sizeof(double), cudaMemcpyHostToDevice);

        Calculate<<<b,a>>>(dev_T,dev_t_old,r,n);

        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
        fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
        }

      
        // cudaMemcpy(T_old, dev_t_old, (n+1)*(n+1)*sizeof(double), cudaMemcpyDeviceToHost);
        
    }
    hipMemcpy(T, dev_t_old, (n+1)*(n+1)*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(&dev_T);
    hipFree(&dev_t_old);
    hipFree(&temp);

    auto stop = high_resolution_clock::now(); 
    auto duration = duration_cast<milliseconds>(stop - start); 

    printf("simulation completed\n");
    printf("Time_Taken :  %d\n",duration.count());

    fstream fout;
    fout.open("soln.dat",ios::out);
    
    for(int i=1;i<n+1;++i)
    {
        for(int j=1;j<n+1;++j)
        {
            fout<<std::scientific<<*(x+i*n+j)<<" "<<*(y+i*n+j)<<" "<<*(T+i*n+j)<<"\n";
        }
    }
    fout.close();
    delete x,y,T,T_old,temp;
    return 0;
}